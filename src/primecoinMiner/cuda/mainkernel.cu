#include "hip/hip_runtime.h"
#include <stdio.h>

#include "mainkernel.h"
#include "mpz.h"    // multiple precision cuda code
#include "cuda_string.h"

//__device__ mpz_t mpzTemp;

#define mpz_clear mpz_destroy
#define mpz_cmp mpz_compare
#define mpz_mul mpz_mult
#define mpz_powm mpz_powmod

//copied constants from prime.h

static const unsigned int nFractionalBits = 24;
static const unsigned int TARGET_FRACTIONAL_MASK = (1u<<nFractionalBits) - 1;
static const unsigned int TARGET_LENGTH_MASK = ~TARGET_FRACTIONAL_MASK;
//static const uint64 nFractionalDifficultyMax = (1llu << (nFractionalBits + 32));
//static const uint64 nFractionalDifficultyMin = (1llu << 32);
//static const uint64 nFractionalDifficultyThreshold = (1llu << (8 + 32));
//static const unsigned int nWorkTransitionRatio = 32;

//end copy

//mpz_div(mpz_t *q, mpz_t *r, mpz_t *n, mpz_t *d)


//extra mpz_functions (quick and dirty...)
__device__ inline void mpz_tdiv_q(mpz_t *ROP, mpz_t *OP1, mpz_t *OP2)
{
    mpz_t mpzTemp;
    mpz_init(&mpzTemp);
    mpz_div(ROP,&mpzTemp,OP1,OP2);
    mpz_destroy(&mpzTemp);
}

__device__ inline void mpz_tdiv_r(mpz_t *ROP, mpz_t *OP1, mpz_t *OP2)
{
    mpz_t mpzTemp;
    mpz_init(&mpzTemp);
    mpz_div(&mpzTemp,ROP,OP1,OP2);
    mpz_destroy(&mpzTemp);
}

__device__ inline unsigned int mpz_get_ui(mpz_t *OP)
{
    return OP->digits[0];
}

//Set product to multiplicator times 2 raised to exponent_of_2. This operation can also be defined as a left shift, exponent_of_2 steps.
__device__ inline void mpz_mul_2exp (mpz_t *product, mpz_t *multiplicator, unsigned long int exponent_of_2)
{
    mpz_t mpzTemp;
    mpz_init(&mpzTemp);
    mpz_set_ui(&mpzTemp,2);
    unsigned int limit = exponent_of_2;
    //well this is ugly
    for(unsigned int i=0; i < limit; i++)
    	mpz_bit_lshift(&mpzTemp);

    mpz_mul(product,multiplicator,&mpzTemp);
    mpz_destroy(&mpzTemp);
}


//end extra mpz

__device__ bool devTargetSetLength(unsigned int nLength, unsigned int& nBits)
{
    if (nLength >= 0xff)
    {
        //printf("[CUDA] error TargetSetLength() : invalid length=%u\n", nLength);
	return false;
    }
    nBits &= TARGET_FRACTIONAL_MASK;
    nBits |= (nLength << nFractionalBits);
    return true;
}

__device__ unsigned int devTargetGetLength(unsigned int nBits)
{
    return ((nBits & TARGET_LENGTH_MASK) >> nFractionalBits);
}

__device__ unsigned int devTargetFromInt(unsigned int nLength)
{
    return (nLength << nFractionalBits);
}

__device__ void devTargetIncrementLength(unsigned int& nBits)
{
    nBits += (1 << nFractionalBits);
}

// Check Fermat probable primality test (2-PRP): 2 ** (n-1) = 1 (mod n)
// true: n is probable prime
// false: n is composite; set fractional length in the nLength output
__device__ bool devFermatProbablePrimalityTest(mpz_t &mpzN, unsigned int& nLength)
{
    mpz_t mpzOne;
    mpz_t mpzTwo;
    //mpz_t mpzEight;

    //TODO: generate constants in a different kernel
    mpz_init(&mpzOne);
    mpz_set_ui(&mpzOne,1);	

    mpz_init(&mpzTwo);
    mpz_set_ui(&mpzTwo,2);

    //mpz_init(&mpzEight);
    //mpz_set_ui(&mpzEight,8);

    // Faster GMP version
    
    //mpz_t mpzN;
    mpz_t mpzE;
    mpz_t mpzR;
    
    //mpz_init_set(mpzN, n.get_mpz_t());

    //e = n -1

    mpz_init(&mpzE);
    mpz_sub(&mpzE, &mpzN, &mpzOne);
    mpz_init(&mpzR);

    //BN_mod_exp(&r, &a, &e, &n);
	// r = 2^(n-1) & n
    mpz_powm(&mpzR, &mpzTwo, &mpzE, &mpzN);

    mpz_destroy(&mpzOne);
    mpz_destroy(&mpzTwo);

    if (mpz_cmp(&mpzR, &mpzOne) == 0)
    {
        mpz_clear(&mpzN);
        mpz_clear(&mpzE);
        mpz_clear(&mpzR);
        
        //printf("[CUDA] Fermat test true\n");
        return true;
    }
    // Failed Fermat test, calculate fractional length
    mpz_sub(&mpzE, &mpzN, &mpzR);
    mpz_mul_2exp(&mpzR, &mpzE, nFractionalBits);
    mpz_tdiv_q(&mpzE, &mpzR, &mpzN);

    unsigned int nFractionalLength = mpz_get_ui(&mpzE);
    mpz_clear(&mpzN);
    mpz_clear(&mpzE);
    mpz_clear(&mpzR);

    if (nFractionalLength >= (1 << nFractionalBits))
    {
	//printf("[CUDA] Error FermatProbablePrimalityTest() : fractional assert : nFractionalLength:%i nFractionalBits:%i\n", nFractionalLength, nFractionalBits);
        return false;
    }

    nLength = (nLength & TARGET_LENGTH_MASK) | nFractionalLength;
    return false;
}

//this version prints results for thread 0
__device__ bool devFermatProbablePrimalityTestWithPrint(mpz_t &mpzN, unsigned int& nLength, unsigned int index)
{
    bool prime = false;

    mpz_t mpzOne;
    mpz_t mpzTwo;
    //mpz_t mpzEight;

    //TODO: generate constants in a different kernel
    mpz_init(&mpzOne);
    mpz_set_ui(&mpzOne,1);	

    mpz_init(&mpzTwo);
    mpz_set_ui(&mpzTwo,2);

    //mpz_init(&mpzEight);
    //mpz_set_ui(&mpzEight,8);

    // Faster GMP version
    
    //mpz_t mpzN;
    mpz_t mpzE;
    mpz_t mpzR;
    
    //mpz_init_set(mpzN, n.get_mpz_t());

    //e = n -1

    mpz_init(&mpzE);
    mpz_sub(&mpzE, &mpzN, &mpzOne);

    if(index == 0)
    {
	//printf("[0] N is: ");
	mpz_print(&mpzN);
	//printf("[0] E is: ");
	mpz_print(&mpzE);
    }

    mpz_init(&mpzR);

    //BN_mod_exp(&r, &a, &e, &n);
    mpz_powm(&mpzR, &mpzTwo, &mpzE, &mpzN);

    if(index == 0)
    {
	//printf("[0] R is: ");
	mpz_print(&mpzR);
    }

    mpz_destroy(&mpzOne);
    mpz_destroy(&mpzTwo);

    if (mpz_cmp(&mpzR, &mpzOne) == 0)
    {
	prime = true;  
	//if(index == 0)      
        	//printf("[0] Fermat test true\n");
    }

    mpz_clear(&mpzN);
    mpz_clear(&mpzE);
    mpz_clear(&mpzR);

    return prime;
    // Failed Fermat test, calculate fractional length
    /*mpz_sub(&mpzE, &mpzN, &mpzR);
    mpz_mul_2exp(&mpzR, &mpzE, nFractionalBits);
    mpz_tdiv_q(&mpzE, &mpzR, &mpzN);

    unsigned int nFractionalLength = mpz_get_ui(&mpzE);
    mpz_clear(&mpzN);
    mpz_clear(&mpzE);
    mpz_clear(&mpzR);

    if (nFractionalLength >= (1 << nFractionalBits))
    {
	if(index==0)
		//printf("[CUDA] Error FermatProbablePrimalityTest() : fractional assert : nFractionalLength:%i nFractionalBits:%i\n", nFractionalLength, nFractionalBits);
        return false;
    }

    nLength = (nLength & TARGET_LENGTH_MASK) | nFractionalLength;
    return false;*/
}

// Test probable primality of n = 2p +/- 1 based on Euler, Lagrange and Lifchitz
// fSophieGermain:
//   true:  n = 2p+1, p prime, aka Cunningham Chain of first kind
//   false: n = 2p-1, p prime, aka Cunningham Chain of second kind
// Return values
//   true: n is probable prime
//   false: n is composite; set fractional length in the nLength output
__device__ bool devEulerLagrangeLifchitzPrimalityTest(mpz_t &mpzN, bool fSophieGermain, unsigned int& nLength)
{

    mpz_t mpzOne;
    mpz_t mpzTwo;
    //mpz_t mpzEight;

    //TODO: generate constants in a different kernel
    mpz_init(&mpzOne);
    mpz_set_ui(&mpzOne,1);	

    mpz_init(&mpzTwo);
    mpz_set_ui(&mpzTwo,2);

    //mpz_init(&mpzEight);
    //mpz_set_ui(&mpzEight,8);

    // Faster GMP version
    //mpz_t mpzN;
    mpz_t mpzE;
    mpz_t mpzR;
    mpz_t temp;

    mpz_init(&temp);    

    mpz_init(&mpzE);
    mpz_sub(&mpzE, &mpzN, &mpzOne);
 
    //mpz_set(&temp,&mpzE);

   //e = (n - 1) >> 1;
    //from hp4: mpz_tdiv_q_2exp(&mpzE, &mpzE, 1);
    mpz_tdiv_q(&temp,&mpzE,&mpzTwo);
    mpz_set(&mpzE,&temp);

    mpz_destroy(&temp);

    mpz_init(&mpzR);
    mpz_powm(&mpzR, &mpzTwo, &mpzE, &mpzN);
   
    //nMod8 = n % 8; 
    //mpz_t mpzNMod8;
    //mpz_init(&mpzNMod8);
    //mpz_tdiv_r(&mpzNMod8,&mpzN, &mpzEight);
    unsigned int nMod8 = mpz_get_ui(&mpzN) % 8;    
    //mpz_destroy(&mpzNMod8);

    bool fPassedTest = false;
    if (fSophieGermain && (nMod8 == 7)) // Euler & Lagrange
        fPassedTest = !mpz_cmp(&mpzR, &mpzOne);
    else if (fSophieGermain && (nMod8 == 3)) // Lifchitz
    {
        mpz_t mpzRplusOne;
        mpz_init(&mpzRplusOne);
        mpz_add(&mpzRplusOne, &mpzR, &mpzOne);
        fPassedTest = !mpz_cmp(&mpzRplusOne, &mpzN);
        mpz_clear(&mpzRplusOne);
    }
    else if ((!fSophieGermain) && (nMod8 == 5)) // Lifchitz
    {
        mpz_t mpzRplusOne;
        mpz_init(&mpzRplusOne);
        mpz_add(&mpzRplusOne, &mpzR, &mpzOne);
        fPassedTest = !mpz_cmp(&mpzRplusOne, &mpzN);
        mpz_clear(&mpzRplusOne);
    }
    else if ((!fSophieGermain) && (nMod8 == 1)) // LifChitz
    {
        fPassedTest = !mpz_cmp(&mpzR, &mpzOne);
    }
    else
    {
        mpz_clear(&mpzN);
        mpz_clear(&mpzE);
        mpz_clear(&mpzR);
        mpz_destroy(&mpzOne);
        mpz_destroy(&mpzTwo);
        //printf("[CUDA] Error in EulerLagrangeLifchitzPrimalityTest() : invalid n %% 8 = %d, %s", nMod8, (fSophieGermain? "first kind" : "second kind"));
        return false;
    }
    
    if (fPassedTest)
    {
        mpz_clear(&mpzN);
        mpz_clear(&mpzE);
        mpz_clear(&mpzR);
	mpz_destroy(&mpzOne);
        mpz_destroy(&mpzTwo);
        return true;
    }
    
    // Failed test, calculate fractional length
    //TODO: RCOPY
    mpz_mul(&mpzE, &mpzR, &mpzR);
    mpz_tdiv_r(&mpzR, &mpzE, &mpzN); // derive Fermat test remainder

    mpz_sub(&mpzE, &mpzN, &mpzR);
    mpz_mul_2exp(&mpzR, &mpzE, nFractionalBits);
    mpz_tdiv_q(&mpzE, &mpzR, &mpzN);

    //Todo: implement mpz_get_ui
    unsigned int nFractionalLength = mpz_get_ui(&mpzE);
    mpz_clear(&mpzN);
    mpz_clear(&mpzE);
    mpz_clear(&mpzR);
    mpz_destroy(&mpzOne);
    mpz_destroy(&mpzTwo);
    
    if (nFractionalLength >= (1 << nFractionalBits))
    {
        //printf("[CUDA] error EulerLagrangeLifchitzPrimalityTest() : fractional assert");
        return false;
    }
    nLength = (nLength & TARGET_LENGTH_MASK) | nFractionalLength;
    return false;
}



// Test Probable Cunningham Chain for: n
// fSophieGermain:
//   true - Test for Cunningham Chain of first kind (n, 2n+1, 4n+3, ...)
//   false - Test for Cunningham Chain of second kind (n, 2n-1, 4n-3, ...)
// Return value:
//   true - Probable Cunningham Chain found (length at least 2)
//   false - Not Cunningham Chain
__device__ bool devProbableCunninghamChainTest(mpz_t &n, bool fSophieGermain, bool fFermatTest, unsigned int& nProbableChainLength)
{
    nProbableChainLength = 0;
    //mpz_class N = n;

    mpz_t N;
    mpz_init(&N);

    mpz_t N_copy;
    mpz_init(&N_copy);

    mpz_set(&N,&n);    

    // Fermat test for n first
    if (!devFermatProbablePrimalityTest(N, nProbableChainLength))
        return false;

    //printf("[CUDA ] N is prime!\n");

    // Euler-Lagrange-Lifchitz test for the following numbers in chain
    while (true)
    {
        devTargetIncrementLength(nProbableChainLength);
	//N = N + N or N *=2
	mpz_set(&N_copy,&N);  
        mpz_mult_u(&N,&N_copy,2);
	// N+ = (fSophieGermain? 1 : (-1))
	mpz_addeq_i(&N,(fSophieGermain? 1 : (-1)));
        if (fFermatTest)
        {
            if (!devFermatProbablePrimalityTest(N, nProbableChainLength))
                break;
        }
        else
        {
            if (!devEulerLagrangeLifchitzPrimalityTest(N, fSophieGermain, nProbableChainLength))
                break;
        }
    }

    mpz_destroy(&N);
    mpz_destroy(&N_copy);

    return (devTargetGetLength(nProbableChainLength) >= 2);
}

// Test probable prime chain for: nOrigin
// Return value:
//   true - Probable prime chain found (one of nChainLength meeting target)
//   false - prime chain too short (none of nChainLength meeting target)
__device__ bool devProbablePrimeChainTest(mpz_t &mpzPrimeChainOrigin, unsigned int nBits, bool fFermatTest, unsigned int& nChainLengthCunningham1, unsigned int& nChainLengthCunningham2, unsigned int& nChainLengthBiTwin)
{
    mpz_t mpzOne;
    mpz_init(&mpzOne);
    mpz_set_ui(&mpzOne,1);

    nChainLengthCunningham1 = 0;
    nChainLengthCunningham2 = 0;
    nChainLengthBiTwin = 0;

    mpz_t mpzPrimeChainOriginMinusOne;
    mpz_t mpzPrimeChainOriginPlusOne;

    mpz_init(&mpzPrimeChainOriginMinusOne);
    mpz_init(&mpzPrimeChainOriginPlusOne);

    mpz_add(&mpzPrimeChainOriginPlusOne,&mpzPrimeChainOrigin,&mpzOne);
    mpz_sub(&mpzPrimeChainOriginMinusOne,&mpzPrimeChainOrigin,&mpzOne);

    // Test for Cunningham Chain of first kind
    devProbableCunninghamChainTest(mpzPrimeChainOriginMinusOne, true, fFermatTest, nChainLengthCunningham1);
    // Test for Cunningham Chain of second kind
    devProbableCunninghamChainTest(mpzPrimeChainOriginPlusOne, false, fFermatTest, nChainLengthCunningham2);
    // Figure out BiTwin Chain length
    // BiTwin Chain allows a single prime at the end for odd length chain
    nChainLengthBiTwin =
        (devTargetGetLength(nChainLengthCunningham1) > devTargetGetLength(nChainLengthCunningham2))?
            (nChainLengthCunningham2 + devTargetFromInt(devTargetGetLength(nChainLengthCunningham2)+1)) :
            (nChainLengthCunningham1 + devTargetFromInt(devTargetGetLength(nChainLengthCunningham1)));

    mpz_destroy(&mpzPrimeChainOriginMinusOne);
    mpz_destroy(&mpzPrimeChainOriginPlusOne);
    mpz_destroy(&mpzOne);

    return (nChainLengthCunningham1 >= nBits || nChainLengthCunningham2 >= nBits || nChainLengthBiTwin >= nBits);
}

__global__ void runCandidateSearch(cudaCandidate *candidates, char *result, unsigned int num_candidates)
{
    mpz_t mpzOne;
    mpz_init(&mpzOne);
    mpz_set_ui(&mpzOne,1);

    /*mpz_t mpzTwo;
    mpz_init(&mpzTwo);
    mpz_set_ui(&mpzTwo,2);*/

    mpz_t mpzN1;
    mpz_init(&mpzN1);

    mpz_t mpzN2;
    mpz_init(&mpzN2);
    //mpz_set_ui(&mpzOne,1);

	unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
	//check bounds
	if (index < num_candidates)
	{
		if(index==0)
		{
			//printf("[0] start! \n");
			//printf("sizeof(struct) = %i\n",sizeof(cudaCandidate));		
		}

		cudaCandidate candidate = candidates[index];

		//if(index==0)
			//printf("[0] candidate is %s\n",candidate.strChainOrigin);

		mpz_t mpzChainOrigin;
		mpz_init(&mpzChainOrigin);
	
		//FIXME: mpz_set_str doesnt work on the device right now
		mpz_set_str(&mpzChainOrigin,candidate.strChainOrigin, index);

		if(index==0)
		{
			//printf("[0] chain origin digits[0]: %x\n", mpzChainOrigin.digits[0]);

			//printf("[0] chain origin:");
			mpz_print(&mpzChainOrigin);
		}

		mpz_add(&mpzN1,&mpzChainOrigin,&mpzOne);
		mpz_sub(&mpzN2,&mpzChainOrigin,&mpzOne);

		unsigned int nLength=0;

		char testresult = 0x00;

		//test for chain of length two
		if(devFermatProbablePrimalityTestWithPrint(mpzN1, nLength, index) || devFermatProbablePrimalityTestWithPrint(mpzN2, nLength, index))
		{
			/*mpz_t mpzN1_copy;
    			mpz_init(&mpzN1_copy);
			mpz_set(&mpzN1_copy,&mpzN1);    

			mpz_t mpzN2_copy;
    			mpz_init(&mpzN2_copy);
			mpz_set(&mpzN2_copy,&mpzN2);  

        		mpz_mult_u(&mpzN1,&mpzN1_copy,2);
			mpz_mult_u(&mpzN2,&mpzN2_copy,2);

			mpz_addeq_i(&mpzN1,1);
			mpz_addeq_i(&mpzN2,-1);

			if(devFermatProbablePrimalityTestWithPrint(mpzN1, nLength, index) || devFermatProbablePrimalityTestWithPrint(mpzN2, nLength, index))
			{*/
				testresult = 0x01;
			//}
		}
		
		result[index] = testresult;
		

		if(index==0)
			//printf("[0] after fermat test\n");


        	/*if(index==0)
			//printf("[0] loaded\n");

		if (devProbablePrimeChainTest(mpzChainOrigin, candidate.blocknBits, false, candidate.nChainLengthCunningham1, candidate.nChainLengthCunningham2, candidate.nChainLengthBiTwin))
		{
			//printf("[CUDA] Found probable chain!\n");
			result[index] = 0x01;
		}else
		{
			result[index] = 0x00;
        	}*/

		mpz_destroy(&mpzChainOrigin);

	}

}

void runCandidateSearchKernel(cudaCandidate *candidates, char *result, unsigned int num_candidates)
{
	//TODO: make gridsize dynamic
	runCandidateSearch<<< 24 , 40>>>(candidates, result, num_candidates);

}
